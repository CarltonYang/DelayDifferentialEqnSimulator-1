#include <cmath>
#include "simulation_cuda.hpp"
#include "cell_param.hpp"
#include "context.hpp"
#include <limits>
#include <iostream>

typedef std::numeric_limits<double> dbl;
using namespace std;

void simulation_cuda::initialize(){
    calc_max_delays(); 
    _delays.update_rates(_parameter_set._delay_sets);
    _rates.update_rates(_parameter_set._rates_base);
    _critValues.update_rates(_parameter_set._critical_values);
    _cl.initialize(4,300,200);
    _baby_cl_cuda.initialize();
}



