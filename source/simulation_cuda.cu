#include "hip/hip_runtime.h"
#include <cmath>
#include "simulation_cuda.hpp"
#include "cell_param.hpp"
#include "model_impl.hpp"
#include "context_impl.hpp"
#include "cuda_context_impl.hpp"
#include <limits>
#include <iostream>

namespace { const char *strerrno(int) { return strerror(errno); } }

template<class T, T Success, const char *(ErrorStr)(T t)>
struct ErrorInfoBase {
  static constexpr bool isSuccess(T t) { return t == Success; }
  static const char *getErrorStr(T t) { return ErrorStr(t); }
};
template<class T> struct ErrorInfo;
template <> struct ErrorInfo<hipError_t> :
  ErrorInfoBase<hipError_t, hipSuccess, hipGetErrorString> {};
template <> struct ErrorInfo<int> :
  ErrorInfoBase<int, 0, strerrno> {};


#define check(RESULT) do {                      \
    check(RESULT, __FILE__, __LINE__);          \
  } while(0)
template<class T>
static void (check)(T result, const char *file, unsigned line) {
  if (ErrorInfo<T>::isSuccess(result)) return;
  std::cerr << file << ":"
            << line << ": "
            << ErrorInfo<T>::getErrorStr(result) << "\n";
  exit(-1);
}

typedef std::numeric_limits<double> dbl;
using namespace std;

void simulation_cuda::initialize(){
    calc_max_delays(); 
    _delays.update_rates(_parameter_set._delay_sets);
    _rates.update_rates(_parameter_set._rates_base);
    _critValues.update_rates(_parameter_set._critical_values);
    _cl.initialize(4,300,200);
    _baby_cl_cuda.initialize();
}

namespace {
    __global__ void cudasim_execute(simulation_cuda& _sim_cu){

        unsigned int k = threadIdx.x;

        // Iterate through each extant cell or context
        if (_sim_cu._width_current == _sim_cu._width_total || k % _sim_cu._width_total <= 10) { // Compute only existing (i.e. already grown)cells
            // Calculate the cell indices at the start of each mRNA and protein's dela
            simulation_cuda::Context c(_sim_cu, k);

            // Perform biological calculations
            c.updateCon(c.calculateRatesOfChange());
        }
        if (threadIdx.x == 0) {
            //printf("Concentration at time step %d, is: %d\n", _sim_cu._j, _sim_cu._baby_cl_cuda[ph11][_sim_cu._j][0]);
            //printf("thread block is %d\n", threadIdx.x);
#if __CUDA_ARCH__ >= 200
    printf("Hi Cuda World");
#endif
            
        }
        if (k==0){
            _sim_cu._j++;
        }
        if (k < NUM_SPECIES) {
            _sim_cu._baby_j[k]++;
        }
    }
} // end namespace


void simulation_cuda::simulate_cuda(RATETYPE sim_time){
    RATETYPE total_step = sim_time/_step_size;

    //Set dimensions
    dim3 dimBlock(_cells_total,1,1); //each cell had own thread

    //dim3 dimGrid(1,1,1); //simulation done on single block
    dim3 dimGrid(1,1,1);

    //hipDeviceSetLimit(hipLimitStackSize, 65536);
    //Run kernel
    for (int i=0;i<total_step;i++){
        cudasim_execute<<<dimGrid, dimBlock>>>(*this);
        //cout<< _j<< " "<<_baby_cl_cuda[ph11][_j][1]<<endl;
    }

    check(hipDeviceSynchronize());
    //convert back to CPU
    if (hipPeekAtLastError() != hipSuccess) {
        cout << "Kernel launch error: " << hipPeekAtLastError() << "\n";
    }

    hipDeviceSynchronize();
}

